﻿#include<iostream>
#include <fstream>
#include <cmath>
#include "GL/glh_glut.h"
#include "hip/hip_runtime.h"
#include ""
#include "simulation.h"
#include <stdio.h>


bool b[256];
int win_w = 600, win_h = 600;
static int level = 1;
using namespace glh;
glut_simple_mouse_interactor object;

float lightpos[4] = { 13, 10.2, 3.2, 0 };
bool wireframe = false;
ImportedModel myModel("cloth2.obj");
ImportedModel myCube("Tcube2.obj");
Mesh* cloth = initial_cloth(myModel);
Mesh* cube = initial_block(myCube);
extern Simulation initial_sim();
extern void drawModel(const Mesh* cloth);
extern void DynamicModel(Simulation& sim);
extern bool push_sim(Simulation& sim, Mesh* cloth);
Simulation sim = initial_sim();
bool isCreatecloth = push_sim(sim, cloth);
bool isCreateblock = push_sim_static(sim, cube);
void initSetting()
{
	b['9'] = false;
	b['d'] = false;
	b['h'] = true;
}
void initOpengl()
{
	glClearColor(1.0, 1.0, 1.0, 1.0);

	// initialize OpenGL lighting
	GLfloat lightPos[] = { 10.0, 10.0, 10.0, 0.0 };
	GLfloat lightAmb[4] = { 0.0, 0.0, 0.0, 1.0 };
	GLfloat lightDiff[4] = { 1.0, 1.0, 1.0, 1.0 };
	GLfloat lightSpec[4] = { 1.0, 1.0, 1.0, 1.0 };

	glLightfv(GL_LIGHT0, GL_POSITION, &lightpos[0]);
	glLightfv(GL_LIGHT0, GL_AMBIENT, lightAmb);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, lightDiff);
	glLightfv(GL_LIGHT0, GL_SPECULAR, lightSpec);

	//glLightModeli(GL_LIGHT_MODEL_COLOR_CONTROL_EXT, GL_SEPARATE_SPECULAR_COLOR_EXT);
	GLfloat black[] = { 0.0, 0.0, 0.0, 1.0 };
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, black);
	glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, GL_TRUE);

	glEnable(GL_LIGHT0);
	glEnable(GL_LIGHTING);
	glEnable(GL_DEPTH_TEST);
}
void begin_window_coords()
{
	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();
	glOrtho(0.0, win_w, 0.0, win_h, -1.0, 1.0);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}
void end_window_coords()
{
	glMatrixMode(GL_PROJECTION);
	glPopMatrix();
	glMatrixMode(GL_MODELVIEW);
}

void drawGround()
{
	glEnable(GL_COLOR_MATERIAL);
	glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);

	glBegin(GL_QUADS);
	glColor3f(1.f, 0.f, 0.f);
	glVertex3f(20, 0, 20);
	glVertex3f(-20, 0, 20);
	glVertex3f(-20, 0, -20);
	glVertex3f(20, 0, -20);
	glEnd();

	glDisable(GL_COLOR_MATERIAL);
}

void draw()
{
	glPushMatrix();
	glScaled(0.4, 0.4, 0.4);
	glRotatef(-90, 1, 0, 0);
	//绘制模型
	drawModel(cloth);
	drawModel(cube);

	glPopMatrix();

	if (b['g'])
		drawGround();
}
void display()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glDisable(GL_DEPTH_TEST);
	glDisable(GL_LIGHTING);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glShadeModel(GL_SMOOTH);

	if (!b['b']) {
		// gradient background
		begin_window_coords();
		glBegin(GL_QUADS);
		glColor3f(0.2, 0.4, 0.8);
		glVertex2f(0.0, 0.0);
		glVertex2f(win_w, 0.0);
		glColor3f(0.05, 0.1, 0.2);
		glVertex2f(win_w, win_h);
		glVertex2f(0, win_h);
		glEnd();
		end_window_coords();
	}

	glMatrixMode(GL_MODELVIEW);

	glLoadIdentity();
	object.apply_transform();

	// draw scene
	if (b['w'])
		glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	else
		glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

	glEnable(GL_LIGHTING);

	glEnable(GL_DEPTH_TEST);
	glEnable(GL_NORMALIZE);

	draw();

	glutSwapBuffers();

}
void idle()
{
	if (b[' '])
		object.trackball.increment_rotation();

	if (b['d'])
	{
		DynamicModel(sim);
	}

	glutPostRedisplay();
}
void resize(int w, int h)
{
	if (h == 0) h = 1;

	glViewport(0, 0, w, h);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)w / (GLfloat)h, 0.1, 500.0);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	object.reshape(w, h);

	win_w = w; win_h = h;
}
void quit()
{
	//quitModel();
	exit(0);
}
void mouse(int button, int state, int x, int y)
{
	object.mouse(button, state, x, y);
}
void motion(int x, int y)
{
	object.motion(x, y);
}
void key(unsigned char k, int x, int y) {
	b[k] = !b[k];
	switch (k) {
	case 'q':
		quit();
		break;
	}
}
void main_menu(int i)
{
	key((unsigned char)i, 0, 0);
}
void initMenu()
{
	glutCreateMenu(main_menu);
	glutAddMenuEntry("Toggle animation [d]", 'd');
	glutAddMenuEntry("Toggle wireframe [w]", 'w');
	glutAddMenuEntry("========================", '=');
	glutAddMenuEntry("Quit/q [esc]", '\033');
	glutAttachMenu(GLUT_RIGHT_BUTTON);
}

int main(int argc, char** argv)
{
	//ImportedModel myModel("shuttle.obj");
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGBA | GLUT_STENCIL);
	glutInitWindowSize(win_w, win_h);
	glutCreateWindow("Cloth Simulator");

	initOpengl();
	//初始化模型
	//initModel(argc, argv, dataPath, stFrame, visPath);

	object.configure_buttons(1);
	object.dolly.dolly[2] = -3;
	object.trackball.incr = rotationf(vec3f(1, 1, 0), 0.05);

	glutDisplayFunc(display);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutIdleFunc(idle);
	glutKeyboardFunc(key);
	glutReshapeFunc(resize);

	initMenu();

	initSetting();

	//启动多线程
	//startMultiThreads(animPath);
	glutMainLoop();

	quit();

	return 0;
}

